#include "hip/hip_runtime.h"
#include "hashsets.h"
#include "crc.h"

#include <stdint.h>
#include <stdio.h>

//TODO check a bunch of hashes, AND SORT THEM BY SIZE!!! or maybe even build a pseudo binary-tree

int main(){
    //Copy hashsets to device
    uint8_t *dh1;
    uint32_t *dh2;
    hipMalloc(&dh1, 256);
    hipMalloc(&dh2, 4*256);
    hipMemcpy(dh1, hashSet1, 256, hipMemcpyHostToDevice);
    hipMemcpy(dh2, hashSet2, 4*256, hipMemcpyHostToDevice);

    uint32_t goal = 0x49541d5a;
    int maxLen = 6;
	for(int len=1;len<=maxLen;len++){
        //Checking 32 chars => max = 32^maxLen
		uint64_t max=32;
		for(int i=0;i<len;i++)
			max*=32;

		uint64_t *d_result;
		hipMalloc(&d_result,sizeof(uint64_t));
		bruteforceCRC32<<<1,1024>>>(d_result,len,max,goal,dh1,dh2);
		hipDeviceSynchronize();

		uint64_t result = 0;
		hipMemcpy(&result, d_result, sizeof(uint64_t), hipMemcpyDeviceToHost);


		if(result!=0){
			char decoded[len];
			decode(decoded,len,result);
			printf("RESULT: %s -> %x\n",decoded,calculateCRC32(decoded,len));
			break;
		}
	}

    hipFree(dh1);
    hipFree(dh2);
	return 0;
}
