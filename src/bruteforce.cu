#include "hip/hip_runtime.h"
#include "hashsets.h"
#include "crc.h"

#include <stdint.h>
#include <stdio.h>

//TODO check a bunch of hashes, AND SORT THEM BY SIZE!!! or maybe even build a pseudo binary-tree

int main(){
    //Copy hashsets to device
    uint8_t *dh1;
    uint32_t *dh2;
    hipMalloc(&dh1, 256);
    hipMalloc(&dh2, 4*256);
    hipMemcpy(dh1, hashSet1, 256, hipMemcpyHostToDevice);
    hipMemcpy(dh2, hashSet2, 4*256, hipMemcpyHostToDevice);

    //Just random hashes, run the program to figure out what they are lol
    uint32_t hashes[] = {0x49541d5a, 0x5ad7f6bc, 0x937db7ec, 0xa988cb16, 0xdf4ac7b9};
    uint32_t goal = 0xdf4ac7b9;

    int maxLen = 7;
	for(int len=1;len<=maxLen;len++){
        //Checking 32 chars => max = 32^maxLen
		uint64_t max=32;
		for(int i=0;i<len;i++)
			max*=32;

		uint64_t *d_result;
		hipMalloc(&d_result,sizeof(uint64_t));
		//bruteforceCRC32<<<1,1024>>>(d_result,len,max,hashes,sizeof(hashes)/4,dh1,dh2);

		bruteforceCRC32<<<1000,1024>>>(d_result,len,max,goal,dh1,dh2);
		hipDeviceSynchronize();

		uint64_t result = 0;
		hipMemcpy(&result, d_result, sizeof(uint64_t), hipMemcpyDeviceToHost);

		if(result!=0){
			char decoded[len];
			decode(decoded,len,result);
			printf("RESULT: %s -> %x\n",decoded,calculateCRC32(decoded,len));
			break;
		}
	}

    hipFree(dh1);
    hipFree(dh2);
	return 0;
}
