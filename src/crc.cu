#include "hip/hip_runtime.h"
#include "crc.h"
#include "hashsets.h"

uint32_t calculateCRC32(char *src, int len){
    uint32_t hash = 0xffffffff;
    for(int i=0;i<len;i++){
        hash = hashSet2[(hash&0xff)^hashSet1[src[i]]]^((hash) >>8);
    }
    return ~hash;
}


int iterative(int max, int len, uint32_t goal){
    for(int i=0;i<max;i++){
        uint32_t hash=0xffffffff;
        for(int j=0;j<len;j++){
            hash = hashSet2[(hash&0xff)^hashSet1[0x41+( (i&(31<<(5*j))) >>(5*j) )]]^((hash) >>8);
        }
        hash = ~hash;
        if(hash == goal){
            return i;
        }
    }
    return 0;
}

//__global__ void bruteforceCRC32(uint64_t *result,int len, uint64_t max,uint32_t *hashes, int lenHashes, uint8_t *hashSet1, uint32_t *hashSet2){
__global__ void bruteforceCRC32(uint64_t *result,int len, uint64_t max,uint32_t goal, uint8_t *hashSet1, uint32_t *hashSet2){
    uint64_t index = blockIdx.x*blockDim.x +threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for(uint64_t i=index;i<max;i+=stride){
        uint32_t hash=0xffffffff;
        for(int j=0;j<len;j++){
            //hashset1 is just 1,2,3,4,5.. 0xff right? FIX THIS FFS!
            //Kirill from the future: nope, see index 64-65
            hash = hashSet2[(hash&0xff)^hashSet1[0x41+( (i&(31<<(5*j))) >>(5*j) )]]^((hash) >>8);
        }
        //TODO: iterate lol
        hash = ~hash;
        if(*result != 0)
            break; // FIXME: this is stupid since it requires a memory read
        if(hash == goal){
            *result=i;
            break;
        }
    }
}

void decode(char* dst, int len, int result){
    for(int i=0;i<len;i++){
        char c = 0x41+((result&(31<<(5*i))) >>(5*i));
        dst[i] = c;
    }
}
